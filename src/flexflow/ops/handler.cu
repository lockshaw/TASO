#include "flexflow/config.h"
#include "flexflow/cuda_helper.h"

using namespace flexflow;

void FFHandler::init() {
  checkCUDA(hipSetDevice(0));
  checkCUDNN(hipdnnCreate(&this->dnn));
  checkCUDA(hipblasCreate(&this->blas));
  this->workSpaceSize = WORK_SPACE_SIZE;
  checkCUDA(hipMalloc(&this->workSpace, this->workSpaceSize));
}
