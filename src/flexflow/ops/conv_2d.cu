/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/model.h"
#include "flexflow/cuda_helper.h"

using namespace flexflow;

Tensor FFModel::conv2d(const Tensor& input,
                       int outChannels,
                       int kernelH, int kernelW,
                       int strideH, int strideW,
                       int paddingH, int paddingW,
                       int groups,
                       ActiMode activation,
                       bool use_bias,
                       const Op* shared_op,
                       char const *name)
{
  assert(input.numDim == 4); /*NCHW*/
  layers.push_back(
    std::unique_ptr<Op>(
      new Conv2D(*this, input, outChannels, kernelH, kernelW,
             strideH, strideW, paddingH, paddingW, groups, activation,
             use_bias, shared_op, name
      )
    )
  );
  return layers.back()->outputs[0];
}

/*
locals[0] = kernel
locals[1] = bias
*/
Conv2D::Conv2D(FFModel& model,
               const Tensor& _input,
               int out_dim,
               int _kernel_h, int _kernel_w,
               int _stride_h, int _stride_w,
               int _padding_h, int _padding_w,
               int _groups,
               ActiMode _activation,
               bool _use_bias,
               const Op* shared_op,
               const char* name)
: Op(model, OP_CONV2D, shared_op, name, _input),
  in_channels(_input.adim[2]), out_channels(out_dim),
  kernel_h(_kernel_h), kernel_w(_kernel_w),
  stride_h(_stride_h), stride_w(_stride_w),
  padding_h(_padding_h), padding_w(_padding_w),
  groups(_groups), activation(_activation), use_bias(_use_bias),
  profiling(model.config.profiling)
{
  assert(_input.numDim == 4);
  // Set output shape
  int input_w = inputs[0].adim[0];
  int input_h = inputs[0].adim[1];
  int output_w = 1 + (input_w + 2 * padding_w - kernel_w) / stride_w;
  int output_h = 1 + (input_h + 2 * padding_h - kernel_h) / stride_h;
  int output_c = out_channels;
  int output_n = inputs[0].adim[3];
  numOutputs = 1;
  outputs[0].numDim = 4;
  outputs[0].adim[0] = output_w;
  outputs[0].adim[1] = output_h;
  outputs[0].adim[2] = output_c;
  outputs[0].adim[3] = output_n;
  weights[0].numDim = 4;
  weights[0].adim[0] = kernel_w;
  weights[0].adim[1] = kernel_h;
  // Require input channels is divisible by groups
  assert(in_channels % groups == 0);
  weights[0].adim[2] = in_channels / groups;
  weights[0].adim[3] = out_channels;
  numWeights = 1;
  if (use_bias) {
    weights[1].numDim = 1;
    weights[1].adim[0] = out_channels;
    numWeights = 2;
  }
}

hipdnnConvolutionFwdAlgo_t
selectConvolutionForwardAlgorithm(hipdnnHandle_t handle,
                                  const hipdnnTensorDescriptor_t xDesc, const void* x,
                                  const hipdnnFilterDescriptor_t wDesc, const void* w,
                                  const hipdnnConvolutionDescriptor_t convDesc,
                                  void* workSpace, size_t workSpaceSize,
                                  const hipdnnTensorDescriptor_t yDesc, void* y);
hipdnnConvolutionBwdFilterAlgo_t
selectConvolutionBackwardFilterAlgorithm(hipdnnHandle_t handle,
                                         const hipdnnTensorDescriptor_t xDesc, const void* x,
                                         const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                         const hipdnnConvolutionDescriptor_t convDesc,
                                         void* workSpace, size_t workSpaceSize,
                                         const hipdnnFilterDescriptor_t dwDesc, void* dw);
hipdnnConvolutionBwdDataAlgo_t
selectConvolutionBackwardDataAlgorithm(hipdnnHandle_t handle,
                                       const hipdnnFilterDescriptor_t wDesc, const void* w,
                                       const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                       const hipdnnConvolutionDescriptor_t convDesc,
                                       void* workSpace, size_t workSpaceSize,
                                       const hipdnnTensorDescriptor_t dxDesc, void* dx);

/*static*/
void Conv2D::forward_kernel(const Conv2DMeta* m,
                            const float* input_ptr,
                            float* output_ptr,
                            const float* filter_ptr,
                            const float* bias_ptr)
{
  float alpha = 1.0f, beta = 0.0f;
  checkCUDNN(hipdnnConvolutionForward(m->handle.dnn, &alpha,
                                     m->inputTensor, input_ptr,
                                     m->filterDesc, filter_ptr,
                                     m->convDesc, m->fwdAlgo,
                                     m->handle.workSpace, m->handle.workSpaceSize,
                                     &beta, m->outputTensor, output_ptr));

  // use_bias == True
  if (bias_ptr != NULL) {
    checkCUDNN(hipdnnAddTensor(m->handle.dnn, &alpha, m->biasTensor,
                              bias_ptr, &alpha, m->outputTensor, output_ptr));
  }
  if (m->relu) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
                                      &alpha, m->outputTensor, output_ptr,
                                      &beta, m->outputTensor, output_ptr));
  }
}

hipdnnConvolutionFwdAlgo_t
selectConvolutionForwardAlgorithm(hipdnnHandle_t handle,
                                  const hipdnnTensorDescriptor_t xDesc, const void* x,
                                  const hipdnnFilterDescriptor_t wDesc, const void* w,
                                  const hipdnnConvolutionDescriptor_t convDesc,
                                  void* workSpace, size_t workSpaceSize,
                                  const hipdnnTensorDescriptor_t yDesc, void* y)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(
      handle, xDesc, x, wDesc, w, convDesc, yDesc, y,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("forwardAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdFilterAlgo_t
selectConvolutionBackwardFilterAlgorithm(hipdnnHandle_t handle,
                                         const hipdnnTensorDescriptor_t xDesc, const void* x,
                                         const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                         const hipdnnConvolutionDescriptor_t convDesc,
                                         void* workSpace, size_t workSpaceSize,
                                         const hipdnnFilterDescriptor_t dwDesc, void* dw)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithmEx(
      handle, xDesc, x, dyDesc, dy, convDesc, dwDesc, dw,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdFilterAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdDataAlgo_t
selectConvolutionBackwardDataAlgorithm(hipdnnHandle_t handle,
                                       const hipdnnFilterDescriptor_t wDesc, const void* w,
                                       const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                       const hipdnnConvolutionDescriptor_t convDesc,
                                       void* workSpace, size_t workSpaceSize,
                                       const hipdnnTensorDescriptor_t dxDesc, void* dx)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithmEx(
      handle, wDesc, w, dyDesc, dy, convDesc, dxDesc, dx,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdDataAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

Conv2DMeta::Conv2DMeta(FFHandler handler)
: OpMeta(handler)
{
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
}

bool Conv2D::measure_compute_time(Simulator* sim,
                                  const ParallelConfig& pc,
                                  float& forward_time,
                                  float& backward_time)
{
  Tensor sub_output, sub_input;
  if(!outputs[0].get_output_sub_tensor(pc, sub_output, OP_CONV2D))
    return false;
  if(!inputs[0].get_input_sub_tensor(pc, sub_input, OP_CONV2D))
    return false;
  int input_w = sub_input.adim[0];
  int input_h = sub_input.adim[1];
  int input_c = sub_input.adim[2];
  int input_n = sub_input.adim[3];
  int output_w = sub_output.adim[0];
  int output_h = sub_output.adim[1];
  int output_c = sub_output.adim[2];
  int output_n = sub_output.adim[3];
  int pad_h = ((output_h - 1) * stride_h + kernel_h - input_h + 1) / 2;
  int pad_w = ((output_w - 1) * stride_w + kernel_w - input_w + 1) / 2;

  Conv2DMeta* m = sim->conv2d_meta;
  m->relu = activation == AC_MODE_RELU;
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, input_n, input_c, input_h, input_w));
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->biasTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, 1, output_c, 1, 1));
  // require input_c is divisible by groups
  assert(input_c % groups == 0);
  checkCUDNN(hipdnnSetFilter4dDescriptor(m->filterDesc, HIPDNN_DATA_FLOAT,
      HIPDNN_TENSOR_NCHW, output_c, input_c / groups, kernel_h, kernel_w));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(m->convDesc, pad_h, pad_w,
      stride_h, stride_w, 1/*dilationH*/, 1/*dilationW*/,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

  checkCUDNN(hipdnnSetConvolutionGroupCount(m->convDesc, groups));
  checkCUDNN(hipdnnSetConvolutionMathType(m->convDesc, HIPDNN_TENSOR_OP_MATH));
  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(m->convDesc,
      m->inputTensor, m->filterDesc, &n, &c, &h, &w));
  assert(n == output_n);
  assert(c == output_c);
  assert(h == output_h);
  assert(w == output_w);
  checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
      HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  // allocate tensors in simulator
  sim->free_all();
  float* input_ptr = (float*)sim->allocate(sub_input.get_volume(), DT_FLOAT);
  assert(input_ptr != NULL);
  float *output_ptr = (float*)sim->allocate(sub_output.get_volume(), DT_FLOAT);
  assert(output_ptr != NULL);
  float* weight_ptr = (float*)sim->allocate((size_t)output_c * input_c * kernel_h * kernel_w / groups, DT_FLOAT);
  assert(weight_ptr != NULL);
  float* bias_ptr = (float*)sim->allocate(output_c, DT_FLOAT);
  assert(bias_ptr != NULL);

  // select forward algorithm
  {
    const int reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(
        m->handle.dnn, m->inputTensor, input_ptr,
        m->filterDesc, weight_ptr, m->convDesc, m->outputTensor, output_ptr,
        reqAlgCnt, &cnt, perfResults,
        m->handle.workSpace, m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    forward_time = perfResults[0].time;
    //for (int i = 0; i < cnt; i++)
    //  printf("conv forward: algo(%d) time(%.4lf)\n", perfResults[i].algo, perfResults[i].time);
  }
  // select forward algorithm
  {
    const int reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithmEx(
        m->handle.dnn, m->inputTensor, input_ptr,
        m->outputTensor, output_ptr, m->convDesc, m->filterDesc, weight_ptr,
        reqAlgCnt, &cnt, perfResults,
        m->handle.workSpace, m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    backward_time = perfResults[0].time;
  }
  {
    const int reqAlgCnt = 8;
    int cnt = 0;
    hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
    checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithmEx(
        m->handle.dnn, m->filterDesc, weight_ptr,
        m->outputTensor, output_ptr, m->convDesc, m->inputTensor, input_ptr,
        reqAlgCnt, &cnt, perfResults,
        m->handle.workSpace, m->handle.workSpaceSize));
    assert(cnt > 0);
    checkCUDNN(perfResults[0].status);
    backward_time += perfResults[0].time;
  }
  printf("[Measure Conv2D] name(%s) input(%d %d %d %d) weight(%d %d %d %d) output(%d %d %d %d) stride(%d %d) padding(%d %d) forward_time(%.4lf) backward_time(%.4lf)\n",
         name,
         input_n, input_c, input_h, input_w,
         output_c, input_c / groups, kernel_h, kernel_w,
         output_n, output_c, output_h, output_w,
         stride_h, stride_w,
         padding_h, padding_w,
         forward_time, backward_time);
  return true;
}
